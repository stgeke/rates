#include "hip/hip_runtime.h"
/*Returns the molar production rate of species */
/*Given P, T, and mole fractions */
extern "C" __global__ void ckwxp_(const int N, double * P, double * T, double * x, double * wdot)
{
    double c[53]; /*temporary storage */
    double rates[53] = {0.0}; /*temporary storage */
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= N)  return;

    const double T_val = T[idx];
    const double PORT = 1.0e6 * P[idx]/(8.314621000e+07 * T_val); /*1.0e6 * P/RT so c goes to SI units */

#pragma unroll
    for (int id = 0; id < 53; ++id) {
        c[id] = x[id*N + idx]*PORT;
    }

    /*convert to chemkin units */
    productionRate(rates, c, T_val);

#pragma unroll
    for (int id = 0; id < 53; ++id) {
        wdot[id*N + idx] = 1.0e-6 * rates[id];
    }
}


